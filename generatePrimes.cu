#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>
using namespace std;
// CUDA kernel. Each thread takes care of one element of c
// If the number at that index is prime then the value in c is set to that value. Value is set to 0 otherwise
__global__ void checkPrimes(unsigned long long int* a, unsigned long long int* c, unsigned long long int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) {
        // Set's value of c to that of a
        c[id] = a[id];
        // Checks if number is zero or 1. These are not prime numbers
        if (a[id] == 0 || a[id] == 1) {
            		c[id] =0;
            	}
            	for(int i = 2; i <= a[id] / 2; i++){
            		if (a[id] % i == 0){
            			c[id] = 0;
                        break;
            		}
            	}
    }
}

int main(int argc, char* argv[])
{
    // File operators
    ofstream primeFile;
    primeFile.open("primes.txt");
    
    unsigned long long int n = 1000000;
    //USER I/O
    cout << "Please enter the number you would like to find primes of: ";
    cin >> n;
    cout << "You have entered the number : " << n<<endl;
    cout << "Beginning the process of finding prime numbers between 0 and " << n << endl;
    size_t bytes = n * sizeof(unsigned long long int);
    // Host input vector
    unsigned long long int* h_a;
    // Host output vector
    unsigned long long int* h_c;
    // Allocate memory for vectors
    h_a = (unsigned long long int*)malloc(bytes);
    h_c = (unsigned long long int*)malloc(bytes);

    // Device input vectors
    unsigned long long int* d_a;
    //Device output vector
    unsigned long long int* d_c;

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_c, bytes);
    
    unsigned long long int i;
    
    // Initialize vectors on host
    cout << "Creating array of values" << endl;
    for (i = 0; i < n; i++) {
        //cout << i << endl;
        h_a[i] = i;
        //cout << i << endl;
    }
    cout << "Finished Generating the list" << endl;

    // Copy host vectors to device
    cout << "Passing memory to GPU" << endl;
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);

    cout << "Beginning the check for primes" << endl;
    // Execute the kernel
    checkPrimes << <gridSize, blockSize >> > (d_a, d_c, n);

    // Copy array back to host
    cout << "Returning results back to CPU" << endl;
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Write prime numbers to file
    cout << "Printing results to file. H_c[23]:  "<<h_c[23] << endl;
    for (i = 0; i < n; i++) {
        if (h_c[i] != 0) {
            primeFile << h_c[i] << "\n";
            cout << h_c[i] << endl;
        }
        
    }
    cout << "Just about done. Beginning to clear Memory" << endl;


    // Clean up memory and close file
    // Release device memory
    hipFree(d_a);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_c);

    cout << "Closing the file" << endl;
    primeFile.close();
    return 0;
}
